#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>

#include "main.cuh"

#define MAX_NUM 22876792454960 // 88888888888888 in base 9

static void parse_input(const char *filename, inst_t **outarr, size_t *outlen);
__global__ void get_max_input(const inst_t *insts, size_t len, i64 *outputs);
__device__ int simulate(const inst_t *insts, size_t len, i64 num);

int main(int argc, char **argv) {
	if(argc <= 1) {
		fprintf(stderr, "error: no filename supplied\n");
		return 1;
	}

	char *filename = argv[1];

	inst_t *insts;
	size_t inst_num;
	parse_input(filename, &insts, &inst_num);

	inst_t *d_insts;
	hipMalloc(&d_insts, inst_num * sizeof(inst_t));
	hipMemcpy(d_insts, insts, inst_num * sizeof(inst_t), hipMemcpyHostToDevice);

	size_t threads_per_block = 256;
	size_t num_blocks = 4096;
	size_t total_threads = threads_per_block * num_blocks;

	i64 *d_outputs;
	hipMalloc(&d_outputs, total_threads * sizeof(i64));

	get_max_input<<<num_blocks, threads_per_block>>>(d_insts, inst_num, d_outputs);

	i64 *outputs = (i64 *) calloc(total_threads, sizeof(i64));
	hipMemcpy(outputs, d_outputs, total_threads * sizeof(i64), hipMemcpyDeviceToHost);

	i64 maximum = 0;
	for(size_t i = 0; i < total_threads; i++) {
		maximum = (outputs[i] > maximum) ? outputs[i] : maximum;
	}

	printf("Finished! maximum = %lld\n", maximum);

	free(outputs);
	hipFree(d_outputs);
	hipFree(d_insts);
	free(insts);

	return 0;
}

__global__ void get_max_input(const inst_t *insts, size_t len, i64 *outputs) {
	size_t base = blockIdx.x * blockDim.x + threadIdx.x;
	size_t stride = blockDim.x * gridDim.x;
	
	for(i64 i = MAX_NUM - base; i >= 0; i -= stride) {
		if(simulate(insts, len, i) == 0) {
			outputs[base] = i;
			return;
		}
	}
	outputs[base] = 0;
}

__device__ int simulate(const inst_t *insts, size_t len, i64 n) {
	return 1; // TODO
}

static opcode_t parse_opcode(const char *line) {
	if(strstr(line, "inp")) {
		return inpop;
	} else if(strstr(line, "add")) {
		return addop;
	} else if(strstr(line, "mul")) {
		return mulop;
	} else if(strstr(line, "div")) {
		return divop;
	} else if(strstr(line, "mod")) {
		return modop;
	} else if(strstr(line, "eql")) {
		return eqlop;
	} else {
		fprintf(stderr, "error: invalid opcode (%s)\n", line);
		exit(1);
	}
}

static reg_t parse_reg(char reg) {
	switch(reg) {
	case 'w':
		return wreg;
	case 'x':
		return xreg;
	case 'y':
		return yreg;
	case 'z':
		return zreg;
	default:
		fprintf(stderr, "error: invalid register (%c)\n", reg);
		exit(1);
	}
}

static inst_t parse_inst(const char *line) {
	inst_t inst;
	inst.opcode = parse_opcode(line);
	inst.reg1 = parse_reg(line[4]);

	if(isalpha(line[6])) {
		inst.reg2 = parse_reg(line[6]);
		inst.regnum = reg;
	} else {
		inst.num2 = atoi(&line[6]);
		inst.regnum = num;
	}

	return inst;
}

static void parse_input(const char *filename, inst_t **outarr, size_t *outlen) {
	FILE *input = fopen(filename, "r");
	*outlen = 0;
	int ch;
	while(EOF != (ch = fgetc(input))) {
		if(ch == '\n') (*outlen)++;
	}

	rewind(input);

	*outarr = (inst_t *)calloc(*outlen, sizeof(inst_t));

	char line[20];
	size_t idx = 0;
	while(NULL != fgets(line, 20, input) && idx < *outlen) {
		(*outarr)[idx] = parse_inst(line);
		idx++;
	}

	if(!feof(input)) {
		fprintf(stderr, "error: didn't read input stream to end\n");
		exit(1);
	}

	fclose(input);
}

